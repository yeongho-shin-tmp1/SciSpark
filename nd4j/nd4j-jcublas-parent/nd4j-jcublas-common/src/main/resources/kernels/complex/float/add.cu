extern "C"

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__global__ void add_float(int n, float *a, float *b, float *sum) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        sum[i] = a[i] + b[i];
    }

}